
#include <hip/hip_runtime.h>
#include <complex>
#include <stdio.h>
#include <math.h>
#include<complex.h>
#include <hip/hip_complex.h>
#define M_PI 3.14159265358979323846
#define coff_size 500
#define threads 8
using namespace std;


//----------------------------------------------------Complex Variables---------------------------------
__managed__ int nsize;


double R=0; 
complex<double> z[coff_size]; 
complex<double> deltaZ[coff_size]; 

__managed__  double deltaZMax;
complex<double> cList2[coff_size];

double epsilon = 1e-6;
//complex<double> QsubJ,fz; 
int max_iter = 800;



//----------------------------------------------------Complex Variables---------------------------------


//--------------------------------------------------Function Prototypes-------------------------------
void durand_kerner(complex<double> cList[],int n); //Prototypes
void calc_theta(int n);
double max_cof(complex<double> cList[],int n);
void printz(complex<double> cList[],int n);
void update_fz(complex<double> cList[],int n,int o);
void printfile(complex<double> cList[],int n,int k,float st);


//--------------------------------------------------Function Prototypes-------------------------------

//--------------------------------------------------GPU Function---------------------------------------

__global__ void calc_delta(hipDoubleComplex *a,hipDoubleComplex *b,hipDoubleComplex *c)
{
    
		int j=threadIdx.x+blockIdx.x*blockDim.x;
 

		hipDoubleComplex QsubJ = make_hipDoubleComplex(1,0);
		hipDoubleComplex mo=make_hipDoubleComplex(-1,0);
	
		for(int i=0;i < nsize;i++) { 
			
			if(i != j)
	  	{ 
					hipDoubleComplex b1=hipCsub(b[j],b[i]);
					QsubJ =hipCmul(QsubJ,b1);
			}
		} 

		hipDoubleComplex fz =make_hipDoubleComplex(1,0);
		for(int k = nsize-1;k >= 0;k--)
	 	{
			//printf("a[%d] = %0.10f + %0.10f*I\n",k,cuCreal(a[k]),cuCimag(a[k]));  
			hipDoubleComplex a1=hipCmul(fz,b[j]);
			fz = hipCadd(a1, a[k]);
		}
		c[j]=hipCdiv(hipCmul(mo,fz),QsubJ);         
}

//--------------------------------------------------GPU Function---------------------------------------

//----------------------------------------------------Main---------------------------------

int main() {
	
  complex<double> cList[coff_size];
 	complex<double> z; 
 	double x,y; //x for real and y for imaginary parts of the coefficient
 	int n=0; //n is number degree of polynomial



 //------Read Coefficients------------------------------------------------

	n=120;
	for(int i=0;i<n;i++ )
	{
			cList[i]=complex<double>(i+1,i+1);
	}

	
	nsize=n;

	cList[n] = complex<double>(1,0); //Store in cList[]

	if(n>=threads)
	durand_kerner(cList,n);
	else
	printf("No of Threads> No of Blocks,hence  program terminated");
	
	
}

//----------------------------------------------------Main----------------------------------------------

//----------------------------------------------------DK Function---------------------------------

void durand_kerner(complex<double> cList[],int n) {

	R = 1 + max_cof(cList,n);  //End Equation 5
	float time = 0,total=0;
	calc_theta(n);
	int k=0;
    hipEvent_t start, stop;
		float elapsedTime;
    hipDoubleComplex *d_a, *d_b,*d_c;
    int size = n*sizeof(cList[0]);

		for(int j=0;j<n;j++)
				{
						z[j]=z[j]+deltaZ[j];
				}


    hipMalloc((void **)&d_a, size);
    hipMemcpy(d_a, &cList2, size, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_b, size);
		hipMalloc((void **)&d_c, size);
		
		hipEventCreate(&start);
    hipEventRecord(start,0);

		for(int i=0;i<max_iter;i++)
		{	
				k+=1;
		 		deltaZMax=0;
				hipMemcpy(d_b, &z, size, hipMemcpyHostToDevice);
				calc_delta<<<n/threads + 1 ,threads>>>(d_a,d_b,d_c);
				hipDeviceSynchronize();
				hipMemcpy(&deltaZ, d_c, size, hipMemcpyDeviceToHost);
			
				for(int j=0;j<n;j++)
				{
						z[j]=z[j]+deltaZ[j];
						if(abs(deltaZ[j]) > deltaZMax)
						{
							deltaZMax = abs(deltaZ[j]);
						}
				}
			if(deltaZMax <= epsilon)
			{ 
					break;   
			}

		}
		hipEventCreate(&stop);
    hipEventRecord(stop,0);
		hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start,stop);
    printf("No of Threads=%d\nNo of iterations=%d\nElapsed time (in seconds): %f\n" ,threads,k,elapsedTime/1000);
		
   printz(cList,n);
	 printfile(cList,n,k,elapsedTime);
      
}

//----------------------------------------------------DK Function----------------------------------------

//----------------------------------------------------Auxiliary Function---------------------------------

void calc_theta(int n) { 
	for(int j=0;j < n;j++) { 
        z[j]=complex<double> (cos(  j*((2*M_PI)/n) )*R,sin(  j*((2*M_PI)/n) )*R);
	} 

}

double max_cof(complex<double> cList[],int n)
{
	double r;
	for(int j=0;j < n;j++) {
			cList2[j]=cList[j];
		if(abs(cList[j]) > R) { 
			r = abs(cList[j]);
		}
	} 	
	return r;
}

void printz(complex<double> cList[],int n)
{
		printf("Final Output:(Note: if the roots repeat then there exist less than n-1 roots for the equation)\n");
		for(int i=0;i < n;i++) {  
                	printf("z[%d] = %0.10f + %0.10f*I\n",i,real(z[i]),imag(z[i]));
                fflush(stdout);
        	}
}

void printfile(complex<double> cList[],int n,int k,float st)
{		
		FILE *fp;  
   		fp = fopen("project_roots.txt", "w");
		fprintf(fp,"Durand Kerner Serial Algorithm:\n");
		fprintf(fp,"Max Iteration=%d\n",k);	
		fprintf(fp,"Time Taken=%f\n",st);
		fprintf(fp,"Final Output:(Note: if the roots repeat then there exist less than n-1 roots for the equation)\n");
		for(int i=0;i < n;i++) {  
                	fprintf(fp,"z[%d] = %0.10f + %0.10f*I\n",i,real(z[i]),imag(z[i]));
                fflush(stdout);
        	}
		fclose(fp);
}
//----------------------------------------------------Auxiliary Function---------------------------------

